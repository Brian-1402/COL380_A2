
#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>
#include <algorithm>
#include <cmath>
#include <chrono>
#include <fstream>
using namespace std;

#define MAXPOOL 0
#define AVGPOOL 1


// Kernel dimensions explained:
// 1) output channel (sets of filters. Each filter corresponds to an output channel)
// 2) input channel (sets of 2d kernel matrices, together called a filter. Each kernel corresponds to each input channel in the input)
// 3) kernel row
// 4) kernel column
__global__ void convLayer(float*** inputMatrix, float*** outputMatrix, float**** kernel, float* bias, int kernel_size, int input_channels, int stride = 1) {
	int i = blockIdx.x * blockDim.x + threadIdx.x; // height of output image
	int j = blockIdx.y * blockDim.y + threadIdx.y; // width of output image
	int k = blockIdx.z * blockDim.z + threadIdx.z; // channel of output image
	float val = 0.0f;

	for (int a = 0; a < input_channels; a++) // Iterate through input channels
		for (int b = 0; b < kernel_size; b++) // Iterate through input rows
			for (int c = 0; c < kernel_size; c++) // Iterate through input columns
				val += inputMatrix[a][i * stride + b][j * stride + c] * kernel[k][a][b][c];

	outputMatrix[k][i][j] = val + bias[k];
}


__global__ void pool(float*** inputMatrix, float*** outputMatrix, int pooldim, int pooltype, int stride = 1) {
	int i = blockIdx.x * blockDim.x + threadIdx.x; // height of output image
	int j = blockIdx.y * blockDim.y + threadIdx.y; // width of output image
	int k = blockIdx.z * blockDim.z + threadIdx.z; // channel of output image
	float val = 0.0f;
	if (pooltype == MAXPOOL)
		val = inputMatrix[k][i * stride][j * stride];

	for (int b = 0; b < pooldim; b++)
		for (int c = 0; c < pooldim; c++)
			if (pooltype == MAXPOOL)
				val = max(val, inputMatrix[k][i * stride + b][j * stride + c]);
			else if (pooltype == AVGPOOL)
				val += inputMatrix[k][i * stride + b][j * stride + c];

	if (pooltype == AVGPOOL)
		val = val / pow(2.0f, pooldim);
	outputMatrix[k][i][j] = val;
}


__global__ void padMatrix(float*** inputMatrix, float*** outputMatix, int padding){
	int i = blockIdx.x * blockDim.x + threadIdx.x; // height of input image
	int j = blockIdx.y * blockDim.y + threadIdx.y; // width of input image
	int k = blockIdx.z * blockDim.z + threadIdx.z; // channel of input image
	outputMatix[k][i + padding][j + padding] = inputMatrix[k][i][j];
}


// Apply ReLU activation to each element of the matrix
__global__ void ReLU(float *** inputMatrix, float *** outputMatrix) {
	// Expects that the data is layed across channel dimension, z
	int k = blockIdx.z * blockDim.z + threadIdx.z;
	outputMatrix[0][0][k] = max(0.0f,inputMatrix[0][0][k]);
}

// softmax value of each element of the vector
__global__ void softmax(float*** inputVector, float*** outputVector) {
	// Expects that the data is layed across channel dimension, z
	int k = blockIdx.z * blockDim.z + threadIdx.z;
	outputVector[0][0][k] = exp(inputVector[0][0][k]);

	__syncthreads();

	// The below code forces a serial computation onto just 1 thread and makes the others wait
	// More efficient way to do this is to use a reduction kernel, or split into two parts
	__shared__ int sum;
	if(threadIdx.x == 0) {
		sum = 0;
		for(int j = 0; j < blockDim.x; j++)
			sum += outputVector[0][0][j];
	}
	__syncthreads();

	outputVector[0][0][k] /= sum;
}
void readFile(std::vector<float>& data, const std::string& filename) {
	std::ifstream file(filename);
	float value;
	while (file >> value) {
		data.push_back(value);
	}
}

#define NUM_FILTERS 0
#define NUM_CHANNELS 1
#define KERNEL_DIM 2

void makeKernel(vector<float>& data, float****& kernel, int* dim) {
	kernel = new float***[dim[NUM_FILTERS]];
	int index = 0;
	for (int i = 0; i < dim[NUM_FILTERS]; ++i) {
		kernel[i] = new float**[dim[NUM_CHANNELS]];
		for (int j = 0; j < dim[NUM_CHANNELS]; ++j) {
			kernel[i][j] = new float*[dim[KERNEL_DIM]];
			for (int k = 0; k < dim[KERNEL_DIM]; ++k) {
				kernel[i][j][k] = new float[dim[KERNEL_DIM]];
				for (int l = 0; l < KERNEL_DIM; ++l)
					kernel[i][j][k][l] = data[index++];
			}
		}
	}
	float**** d_kernel;
	hipMalloc(&d_kernel, dim[NUM_FILTERS] * sizeof(float**));
	hipMemcpy(d_kernel, kernel, dim[NUM_FILTERS] * sizeof(float**), hipMemcpyHostToDevice);
	kernel = d_kernel;
}

void makeBias(std::vector<float>& data, float*& bias, int* dim) {
	bias = new float[dim[NUM_FILTERS]];
	for (int i = 0; i < dim[NUM_FILTERS]; ++i) {
		bias[i] = data[data.size() - dim[NUM_FILTERS] + i];
	}
	float* d_bias;
	hipMalloc(&d_bias, dim[NUM_FILTERS] * sizeof(float));
	hipMemcpy(d_bias, bias, dim[NUM_FILTERS] * sizeof(float), hipMemcpyHostToDevice);
	bias = d_bias;
}

/*

// Usage of the read functions
int main() {
	std::vector<float> data;
	readFile(data, "conv1.txt");

	float**** kernel;
	to4D(data, kernel);

	float* bias;
	to1D(data, bias);
}

*/


struct weights_struct {
	float**** conv1_kernel;
	float* conv1_bias;
	float**** conv2_kernel;
	float* conv2_bias;
	float**** fc1_kernel;
	float* fc1_bias;
	float**** fc2_kernel;
	float* fc2_bias;
};

// kernel dim: (filters, channels, kernel_dim)
struct weights_struct prep_weights() {
	weights_struct weights;
	std::vector<float> data;
	readFile(data, "./weights/conv1.txt");
	int dim[3] = {20, 1, 5};
	makeKernel(data, weights.conv1_kernel, dim);
	makeBias(data, weights.conv1_bias, dim);
	readFile(data, "./weights/conv2.txt");
	dim[0] = 50;
	dim[1] = 20;
	dim[2] = 5;
	makeKernel(data, weights.conv2_kernel, dim);
	makeBias(data, weights.conv2_bias, dim);
	readFile(data, "./weights/fc1.txt");
	dim[0] = 500;
	dim[1] = 50;
	dim[2] = 4;
	makeKernel(data, weights.fc1_kernel, dim);
	makeBias(data, weights.fc1_bias, dim);
	readFile(data, "./weights/fc2.txt");
	dim[0] = 10;
	dim[1] = 500;
	dim[2] = 1;
	makeKernel(data, weights.fc2_kernel, dim);
	makeBias(data, weights.fc2_bias, dim);
	return weights;
}

//Requires arguments to already be in CUDA memory
void forward_prop(float ***inputImage, float ***outputVector, weights_struct weights, hipStream_t stream){
	float *** c1_out, *** p1_out, *** c2_out, *** p2_out, *** fc1_out, ***fc1_relu_out, *** fc2_out, *** fc2_softmax_out;
	// dimensions: (height, width, output_channels)
	hipMalloc(&c1_out, 24 * 24 * 20 * sizeof(float));
	hipMalloc(&p1_out, 12 * 12 * 20 * sizeof(float));
	hipMalloc(&c2_out, 8 * 8 * 50 * sizeof(float));
	hipMalloc(&p2_out, 4 * 4 * 50 * sizeof(float));
	hipMalloc(&fc1_out, 1 * 1 * 500 * sizeof(float));
	hipMalloc(&fc1_relu_out, 1 * 1 * 500 * sizeof(float));
	hipMalloc(&fc2_out, 1 * 1 * 10 * sizeof(float));
	// cudaMalloc(&fc2_softmax_out, 1 * 1 * 10 * sizeof(float));
	// C1: 45 blocks, 256 threads per block
	convLayer<<<dim3(3,3,5), dim3(8,8,4), 0, stream>>>(inputImage, c1_out, weights.conv1_kernel, weights.conv1_bias, 5, 1);
	// P1: 16 blocks, 180 threads per block
	pool<<<dim3(2,2,4), dim3(6,6,5), 0, stream>>>(c1_out, p1_out, 2, MAXPOOL, 2);
	// C2: 20 blocks, 160 threads per block
	convLayer<<<dim3(2,2,5), dim3(4,4,10), 0, stream>>>(p1_out, c2_out, weights.conv1_kernel, weights.conv1_bias, 5, 1);
	// P2: 5 blocks, 160 threads per block
	pool<<<dim3(1,1,5), dim3(4,4,10), 0, stream>>>(c2_out, p2_out, 2, MAXPOOL, 2);
	// FC1: 4 blocks, 125 threads per block
	convLayer<<<dim3(1,1,4), dim3(1,1,125), 0, stream>>>(p2_out, fc1_out, weights.fc1_kernel, weights.fc1_bias, 4, 1);
	// Relu1: 4 blocks, 125 threads per block
	ReLU<<<dim3(1,1,4), dim3(1,1,125), 0, stream>>>(fc1_out, fc1_relu_out);
	// FC2: 1 block, 10 threads per block
	convLayer<<<dim3(1,1,1), dim3(1,1,10), 0, stream>>>(fc1_relu_out, fc2_out, weights.fc2_kernel, weights.fc2_bias, 1, 1);
	// Softmax: 1 block, 10 threads per block
	softmax<<<dim3(1,1,1), dim3(1,1,10), 0, stream>>>(fc2_out, outputVector);
}

int main() {
	// vector<float> data;
	// readFile(data, "./weights/conv2.txt");
	// for (int i = 0; i < data.size(); i++)
	//	cout << data[i] << endl;
}
